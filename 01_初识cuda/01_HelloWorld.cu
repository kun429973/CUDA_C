
#include <hip/hip_runtime.h>
#include <stdio.h>

__host__ __device__ void hello_from_hostANDdevice() {
    printf("Hello World from hostANDdevice!\n");
}


__device__ void hello_from_device() {
    printf("Hello World from device!\n");
}

__global__ void hello_from_global() {
    printf("Hello World from global!\n");
    hello_from_device();
    hello_from_hostANDdevice();
}

int main(void) {
    hello_from_global<<<1, 1>>>();
    hipDeviceSynchronize(); 
    printf("__host__端 log:\n");
    hello_from_hostANDdevice();
    
}
