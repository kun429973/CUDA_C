
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void hello_from_device() {
    printf("Hello World from device!\n");
}

__host__ __device__ void hello_from_hostANDdevice() {
    printf("Hello World from hostANDdevice!\n");
#ifdef __CUDA_ARCH__
    printf("__CUDA_ARCH__: \n");
    hello_from_device();
#endif
}

__global__ void hello_from_global() {
    printf("Hello World from global!\n");
    hello_from_device();
    hello_from_hostANDdevice();
}

int main(void) {
    hello_from_global<<<1, 1>>>();
    hipDeviceSynchronize();
    printf("__host__端 log:\n");
    
}


